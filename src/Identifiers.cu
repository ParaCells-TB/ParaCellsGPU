#include "Identifiers.cuh"

#include <cstdlib>
#include <cstring>
#include "hip/hip_runtime.h"

#include "BuiltInKernels.cuh"
#include "ParaCellsError.cuh"

Identifiers::Identifiers(int maxIdentifiersNum)
{
	//Size initialization
	this->maxIdentifiersNum = maxIdentifiersNum;
	this->currentIdentifiersNum = 0;

	//Unpushed data flag initialization
	this->hasUnpushedChangesInHost = 0;
	this->hasUnpulledChangesInDevice = 0;

	//CPU identifiers memory allocation
	h_identifiers = (char**)malloc(sizeof(char *)*maxIdentifiersNum);

	//GPU identifiers memory allocation
	CUDACall(hipMalloc((void**)&d_identifiers, sizeof(char *)*maxIdentifiersNum));
}

char **Identifiers::getHostIdentifiers()
{
	return h_identifiers;
}

char **Identifiers::getDeviceIdentifiers()
{
	return d_identifiers;
}

int Identifiers::getMaxIdentifiersNum()
{
	return maxIdentifiersNum;
}

void Identifiers::setCurrentIdentifiersNum(int value)
{
	if (value > maxIdentifiersNum)
	{
		raiseError("The number of identifiers exceeded!");
	}

	currentIdentifiersNum = value;
}

int Identifiers::getCurrentIdentifiersNum()
{
	return currentIdentifiersNum;
}

void Identifiers::addIdentifier(int index, const char *identifierName)
{
	if (index >= maxIdentifiersNum)
	{
		raiseError("The index of identifier exceeded!");
	}

	setCurrentIdentifiersNum(currentIdentifiersNum + 1);

	int len = strlen(identifierName);

	//CPU memory allocation
	h_identifiers[index] = (char*)malloc(sizeof(char)*(len+1));

	//CPU string copy
	strcpy(h_identifiers[index], identifierName);

	//GPU string alloc&copy by kernel
	char *d_identifierName;
	CUDACall(hipMalloc((void **)&d_identifierName, sizeof(char)*(len + 1)));
	CUDACall(hipMemcpy(d_identifierName, identifierName, sizeof(char)*(len + 1), hipMemcpyHostToDevice));
	addIdentifierKernel<<<1, 1>>>(d_identifiers, index, d_identifierName);
	CUDACall(hipGetLastError());
	CUDACall(hipDeviceSynchronize());
}

int Identifiers::findIdentifier(const char *identifierName)
{
	for (int i = 0; i < currentIdentifiersNum; i++)
	{
		if (strcmp(identifierName, h_identifiers[i]) == 0)
		{
			//Found
			return i;
		}
	}

	//Not found
	return -1;
}

Identifiers::~Identifiers()
{
	//free identifiers' memory on host
	for (int i = 0; i < currentIdentifiersNum; i++)
	{
		free(h_identifiers[i]);
	}
	free(h_identifiers);

	//free identifiers' memory on device
	freeIdentifierKernel<<<1, 1>>>(d_identifiers, currentIdentifiersNum);
	CUDACall(hipGetLastError());
	CUDACall(hipDeviceSynchronize());
	hipFree(d_identifiers);
}